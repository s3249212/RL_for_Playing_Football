#include "hip/hip_runtime.h"
#include <cstdlib>
#include <vector>
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <iostream>
#include <iomanip>

#define CUDA_CALL(f) { \
  hipError_t err = (f); \
  if (err != hipSuccess) { \
    std::cout \
        << "    Error occurred: " << err << std::endl; \
    std::exit(1); \
  } \
}

#define CUDNN_CALL(f) { \
  hipdnnStatus_t err = (f); \
  if (err != HIPDNN_STATUS_SUCCESS) { \
    std::cout \
        << "    Error occurred: " << err << std::endl; \
    std::exit(1); \
  } \
}

__global__ void dev_const(float *px, float k) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  px[tid] = k;
}

__global__ void dev_iota(float *px) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  px[tid] = tid;
}

void print(const float *data, int n, int c, int h, int w) {
    std::vector<float> buffer(1 << 20);
    CUDA_CALL(hipMemcpy(
          buffer.data(), data,
          n * c * h * w * sizeof(float),
          hipMemcpyDeviceToHost));
    int a = 0;
    for (int i = 0; i < n; ++i) {
      for (int j = 0; j < c; ++j) {
        std::cout << "n=" << i << ", c=" << j << ":" << std::endl;
        for (int k = 0; k < h; ++k) {
          for (int l = 0; l < w; ++l) {
            std::cout << std::setw(4) << std::right << buffer[a];
            ++a;
          }
          std::cout << std::endl;
        }
      }
    }
    std::cout << std::endl;
  }

int main()
{
    hipdnnHandle_t cudnn;
    CUDNN_CALL(hipdnnCreate(&cudnn));

    // input
    const int in_n = 1; //batch size
    const int in_c = 1; //number of feature maps(channel)
    const int in_h = 5; //image height
    const int in_w = 5; //image width

    hipdnnTensorDescriptor_t in_desc;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&in_desc));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(
            in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
            in_n, in_c, in_h, in_w));

    float *in_data;
    CUDA_CALL(hipMalloc(
            &in_data, in_n * in_c * in_h * in_w * sizeof(float)));

    // filter
    const int filt_k = 1; //the number of the output channel
    const int filt_c = 1; //the number of the channel
    const int filt_h = 2; //filter height
    const int filt_w = 2; //filter width

    hipdnnFilterDescriptor_t filt_desc;
    CUDNN_CALL(hipdnnCreateFilterDescriptor(&filt_desc));
    CUDNN_CALL(hipdnnSetFilter4dDescriptor(
            filt_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
            filt_k, filt_c, filt_h, filt_w));

    float *filt_data;
    CUDA_CALL(hipMalloc(
        &filt_data, filt_k * filt_c * filt_h * filt_w * sizeof(float)));

    // convolution
    const int pad_h = 1; //padding
    const int pad_w = 1;
    const int str_h = 1; //stride
    const int str_w = 1;
    const int dil_h = 1; //dilation (space between the cells)
    const int dil_w = 1;

    hipdnnConvolutionDescriptor_t conv_desc;
    CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
    CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
            conv_desc,
            pad_h, pad_w, str_h, str_w, dil_h, dil_w,
            HIPDNN_CONVOLUTION, HIPDNN_DATA_FLOAT));

    // output
    int out_n;
    int out_c;
    int out_h;
    int out_w;
    
    CUDNN_CALL(hipdnnGetConvolution2dForwardOutputDim(
            conv_desc, in_desc, filt_desc,
            &out_n, &out_c, &out_h, &out_w));

    hipdnnTensorDescriptor_t out_desc;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(
            out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
            out_n, out_c, out_h, out_w));

    float *out_data;
    CUDA_CALL(hipMalloc(
            &out_data, out_n * out_c * out_h * out_w * sizeof(float)));

    // algorithm
    hipdnnConvolutionFwdAlgo_t algo;
    CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(
            cudnn,
            in_desc, filt_desc, conv_desc, out_desc,
            HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));

    // workspace
    size_t ws_size;
    CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
            cudnn, in_desc, filt_desc, conv_desc, out_desc, algo, &ws_size));

    float *ws_data;
    CUDA_CALL(hipMalloc(&ws_data, ws_size));

    float alpha = 1.f;
    float beta = 0.f;
    dev_iota<<<in_w * in_h, in_n * in_c>>>(in_data);
    dev_const<<<filt_w * filt_h, filt_k * filt_c>>>(filt_data, 1.f);
    CUDNN_CALL(hipdnnConvolutionForward(
            cudnn,
            &alpha, in_desc, in_data, filt_desc, filt_data,
            conv_desc, algo, ws_data, ws_size,
            &beta, out_desc, out_data));

    // results
    std::cout << "in_data:" << std::endl;
    print(in_data, in_n, in_c, in_h, in_w);
    
    std::cout << "filt_data:" << std::endl;
    print(filt_data, filt_k, filt_c, filt_h, filt_w);
    
    std::cout << "out_data:" << std::endl;
    print(out_data, out_n, out_c, out_h, out_w);



    // finalizing
    CUDA_CALL(hipFree(ws_data));
    CUDA_CALL(hipFree(out_data));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(out_desc));
    CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_desc));
    CUDA_CALL(hipFree(filt_data));
    CUDNN_CALL(hipdnnDestroyFilterDescriptor(filt_desc));
    CUDA_CALL(hipFree(in_data));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(in_desc));
    CUDNN_CALL(hipdnnDestroy(cudnn));

}